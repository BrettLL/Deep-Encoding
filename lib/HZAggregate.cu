#include "hip/hip_runtime.h"
/*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
 * Created by: Hang Zhang
 * ECE Department, Rutgers University
 * Email: zhang.hang@rutgers.edu
 * Copyright (c) 2016
 *
 * Feel free to reuse and distribute this software for research or 
 * non-profit purpose, subject to the following conditions:
 *  1. The code must retain the above copyright notice, this list of
 *     conditions.
 *  2. Original authors' names are not deleted.
 *  3. The authors' names are not used to endorse or promote products
 *      derived from this software 
 *+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
 */
__global__ void HZAggregate_Forward_kernel (
	THCDeviceTensor<real, 3> E,
	THCDeviceTensor<real, 3> A,
	THCDeviceTensor<real, 4> R)
{
  /* declarations of the variables */
  int b, k, d, i, N;
	real sum;
  /* Get the index and channels */ 
  b = blockIdx.z;
  d = blockIdx.x * blockDim.x + threadIdx.x;
  k = blockIdx.y * blockDim.y + threadIdx.y;
	N = A.getSize(1);
	/* boundary check for output */
	sum = 0;
	if (d >= E.getSize(2) || k >= E.getSize(1))	return;
	/* main operation */
	for(i=0; i<N; i++) {
		sum += A[b][i][k].ldg() * R[b][i][k][d].ldg();
	}
	E[b][k][d] = sum;
}

void HZAggregate_Forward(THCState *state, THCTensor *E_, THCTensor *A_,
							THCTensor *R_)
/*
 * mapping the image pixels based on the lookuptable
 */
{
	/* Check the GPU index */
	HZENCODING_assertSameGPU(state, 3, E_, A_, R_);
	/* Device tensors */
	THCDeviceTensor<real, 3> E = devicetensor<3>(state, E_);
	THCDeviceTensor<real, 3> A = devicetensor<3>(state, A_);
	THCDeviceTensor<real, 4> R = devicetensor<4>(state, R_);
	/* kernel function */
	hipStream_t stream = THCState_getCurrentStream(state);
	dim3 threads(16, 16);
	dim3 blocks(E.getSize(2)/16+1, E.getSize(1)/16+1, 
							E.getSize(0));
	HZAggregate_Forward_kernel<<<blocks, threads, 0, stream>>>(E, A, R);
	THCudaCheck(hipGetLastError());
}

__global__ void HZAggregate_BackwardA_kernel (
	THCDeviceTensor<real, 3> G,
	THCDeviceTensor<real, 3> L,
	THCDeviceTensor<real, 4> R)
{
  /* declarations of the variables */
  int b, k, d, i, D;
	real sum;
  /* Get the index and channels */ 
  b = blockIdx.z;
  k = blockIdx.x * blockDim.x + threadIdx.x;
  i = blockIdx.y * blockDim.y + threadIdx.y;
	D = L.getSize(2);
	/* boundary check for output */
	if (k >= G.getSize(2) || i >= G.getSize(1))	return;
	/* main operation */
	sum = 0;
	for(d=0; d<D; d++) {
		sum += L[b][k][d].ldg() * R[b][i][k][d].ldg();
	}
	G[b][i][k] = sum;
}

void HZAggregate_BackwardA(THCState *state, THCTensor *G_, THCTensor *L_,
							THCTensor *R_)
/*
 * mapping the image pixels based on the lookuptable
 */
{
	/* Check the GPU index */
	HZENCODING_assertSameGPU(state, 3, G_, L_, R_);
	/* Device tensors */
	THCDeviceTensor<real, 3> G = devicetensor<3>(state, G_);
	THCDeviceTensor<real, 3> L = devicetensor<3>(state, L_);
	THCDeviceTensor<real, 4> R = devicetensor<4>(state, R_);
	/* kernel function */
	hipStream_t stream = THCState_getCurrentStream(state);
	dim3 threads(16, 16);
	dim3 blocks(G.getSize(2)/16+1, G.getSize(1)/16+1, 
							G.getSize(0));
	HZAggregate_BackwardA_kernel<<<blocks, threads, 0, stream>>>(G, L, R);
	THCudaCheck(hipGetLastError());
}
