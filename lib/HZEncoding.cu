#include "hip/hip_runtime.h"
/*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
 * Created by: Hang Zhang
 * ECE Department, Rutgers University
 * Email: zhang.hang@rutgers.edu
 * Copyright (c) 2016
 *
 * Feel free to reuse and distribute this software for research or 
 * non-profit purpose, subject to the following conditions:
 *  1. The code must retain the above copyright notice, this list of
 *     conditions.
 *  2. Original authors' names are not deleted.
 *  3. The authors' names are not used to endorse or promote products
 *      derived from this software 
 *+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
 */
__global__ void HZEncoding_ForwardF_kernel (
	THCDeviceTensor<real, 3> F,
	THCDeviceTensor<real, 2> C,
	THCDeviceTensor<real, 1> s,
	THCDeviceTensor<real, 3> X)
{
  /* declarations of the variables */
  int b, k, i, d, D;
	real sum;
  /* Get the index and channels */ 
  b = blockIdx.z;
  k = blockIdx.x * blockDim.x + threadIdx.x;
  i = blockIdx.y * blockDim.y + threadIdx.y;
	D = C.getSize(1);
	/* boundary check for output */
	if (k >= F.getSize(2) || i >= F.getSize(1))	return;
	/* main operation */
	sum = 0;
	for (d=0; d<D; d++) {
		sum += (X[b][i][d].ldg() - C[k][d].ldg()) * (X[b][i][d].ldg() - C[k][d].ldg());
	}
	F[b][i][k] = exp(-s[k] * sum);
}

void HZEncoding_ForwardF(THCState *state, THCTensor *F_, THCTensor *C_,
							THCTensor *s_, THCTensor *X_)
/*
 * mapping the image pixels based on the lookuptable
 */
{
	/* Check the GPU index */
	HZENCODING_assertSameGPU(state, 3, F_, C_, s_, X_);
	/* Device tensors */
	THCDeviceTensor<real, 3> F = devicetensor<3>(state, F_);
	THCDeviceTensor<real, 2> C = devicetensor<2>(state, C_);
	THCDeviceTensor<real, 1> s = devicetensor<1>(state, s_);
	THCDeviceTensor<real, 3> X = devicetensor<3>(state, X_);
	/* kernel function */
	hipStream_t stream = THCState_getCurrentStream(state);
	dim3 threads(16, 16);
	dim3 blocks(F.getSize(2)/16+1, F.getSize(1)/16+1, F.getSize(0));
	HZEncoding_ForwardF_kernel<<<blocks, threads, 0, stream>>>(F, C, s, X);
	THCudaCheck(hipGetLastError());
}

