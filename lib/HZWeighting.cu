#include "hip/hip_runtime.h"
/*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
 * Created by: Hang Zhang
 * ECE Department, Rutgers University
 * Email: zhang.hang@rutgers.edu
 * Copyright (c) 2016
 *
 * Free to reuse and distribute this software for research or 
 * non-profit purpose, subject to the following conditions:
 *  1. The code must retain the above copyright notice, this list of
 *     conditions.
 *  2. Original authors' names are not deleted.
 *  3. The authors' names are not used to endorse or promote products
 *      derived from this software 
 *+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++
 */
__global__ void HZWeighting_UpdateParams_kernel (
	THCDeviceTensor<real, 2> G,
	THCDeviceTensor<real, 3> L,
	THCDeviceTensor<real, 3> D)
{
  /* declarations of the variables */
  int b, k, i, N;
	real sum;
  /* Get the index and channels */ 
  b = blockIdx.y;
  k = blockIdx.x * blockDim.x + threadIdx.x;
	N = L.getSize(1);
	/* boundary check for output */
	if (k >= G.getSize(1))	return;
	/* main operation */
	sum = 0;
	for(i=0; i<N; i++) {
		sum += L[b][i][k].ldg() * D[b][i][k].ldg();
	}
	G[b][k] = isnan(sum) ? 1e-6 : sum;
}

void HZWeighting_UpdateParams(THCState *state, THCTensor *G_, THCTensor *L_,
							THCTensor *D_)
/*
 * mapping the image pixels based on the lookuptable
 */
{
	/* Check the GPU index */
	HZENCODING_assertSameGPU(state, 3, G_, L_, D_);
	/* Device tensors */
	THCDeviceTensor<real, 2> G = devicetensor<2>(state, G_);
	THCDeviceTensor<real, 3> L = devicetensor<3>(state, L_);
	THCDeviceTensor<real, 3> D = devicetensor<3>(state, D_);
	/* kernel function */
	hipStream_t stream = THCState_getCurrentStream(state);
	dim3 threads(16);
	dim3 blocks(G.getSize(1)/16+1,G.getSize(0));
	HZWeighting_UpdateParams_kernel<<<blocks, threads, 0, stream>>>(G, L, D);
	THCudaCheck(hipGetLastError());
}

__global__ void HZWeighting_BatchRowWeighing_kernel (
	THCDeviceTensor<real, 3> G,
	THCDeviceTensor<real, 2> W,
	THCDeviceTensor<real, 3> L)
{
  /* declarations of the variables */
  int b, k, d;
	real output;
  /* Get the index and channels */ 
  b = blockIdx.z;
  d = blockIdx.x * blockDim.x + threadIdx.x;
  k = blockIdx.y * blockDim.y + threadIdx.y;
	/* boundary check for output */
	if (k >= G.getSize(1) || d >= G.getSize(2))	return;
	/* main operation */
	output = L[b][k][d].ldg() * W[b][k].ldg();
	G[b][k][d] = isnan(output) ? 1e-16: output;
}

void HZWeighting_BatchRowWeighting(THCState *state, THCTensor *G_, THCTensor *W_,
							THCTensor *L_)
/*
 * mapping the image pixels based on the lookuptable
 */
{
	/* Check the GPU index */
	HZENCODING_assertSameGPU(state, 3, G_, W_, L_);
	/* Device tensors */
	THCDeviceTensor<real, 3> G = devicetensor<3>(state, G_);
	THCDeviceTensor<real, 2> W = devicetensor<2>(state, W_);
	THCDeviceTensor<real, 3> L = devicetensor<3>(state, L_);
	/* kernel function */
	hipStream_t stream = THCState_getCurrentStream(state);
	dim3 threads(16,16);
	dim3 blocks(G.getSize(2)/16+1, G.getSize(1)/16+1, G.getSize(0));
	HZWeighting_BatchRowWeighing_kernel<<<blocks, threads, 0, stream>>>(G, W, L);
	THCudaCheck(hipGetLastError());
}

